#include <hip/hip_runtime.h>

#include "my_utils.hpp"
#include "my_cuda_utils.hpp"
#include "my_cufft_utils.hpp"

#include "device_allocator.hpp"
#include "managed_allocator_host.hpp"
#include "managed_allocator_global.hpp"

#include "NormAutocorrGPU.cuh"

#include "norm_autocorr_kernel.cuh"

void NormAutocorrGPU::run() {
   try {
      hipError_t cerror = hipSuccess;
      int num_shared_bytes = 0;
      int num_blocks = (adjusted_num_samples + threads_per_block - 1) / threads_per_block;

      dout << __func__ << "(): num_samples is " << num_samples << "\n"; 
      dout << __func__ << "(): threads_per_block is " << threads_per_block << "\n"; 
      dout << __func__ << "(): adjusted_num_samples is " << adjusted_num_samples << "\n"; 
      dout << __func__ << "(): num_blocks is " << num_blocks << "\n"; 

      if ( debug ) {
         print_cufftComplexes( exp_samples_d16, num_samples, "Expected Samples D16: ", " ", "\n" ); 
         print_cufftComplexes( exp_conj_sqrs, num_samples, "Expected Conjugate Squares: ", " ", "\n" );
         print_cufftComplexes( exp_conj_sqr_means, num_samples, "Expected Conjugate Square Means: ", " ", "\n" );
         print_vals( exp_conj_sqr_mean_mags, num_samples, "Expected Conjugate Square Mean Mags: ", " ", "\n" ); 
         print_vals( exp_mag_sqrs, num_samples, "Expected Magnitude Squares: ", " ", "\n" ); 
         print_vals( exp_mag_sqr_means, num_samples, "Expected Magnitude Square Means: ", " ", "\n" );
         print_vals( exp_norms, num_samples, "Expected Norms: ", " ", "\n" ); 
      }
      
      float gpu_milliseconds = 0.f;
      Time_Point start = Steady_Clock::now();
      
      hipStreamAttachMemAsync( *(stream_ptr.get()), samples.data(), 0, hipMemAttachGlobal );

      norm_autocorr_kernel<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
         norms.data(), 
         mag_sqr_means.data(), 
         mag_sqrs.data(), 
         conj_sqr_mean_mags.data(), 
         conj_sqr_means.data(), 
         conj_sqrs.data(), 
         samples_d16.data(), 
         samples.data(),
         conj_sqrs_window_size,
         mag_sqrs_window_size,
         num_samples 
      );

      //// Prefetch fspecs from the GPU
      hipStreamAttachMemAsync( *(stream_ptr.get()), norms.data(), 0, hipMemAttachHost );   
      
      //try_cuda_func_throw( cerror, hipStreamSynchronize( *(stream_ptr.get())  ) );
      try_cuda_func_throw( cerror, hipDeviceSynchronize() );
      
      Duration_ms duration_ms = Steady_Clock::now() - start;
      gpu_milliseconds = duration_ms.count();

      float max_diff = 1;
      bool all_close = false;
      if ( debug ) {
         std::cout << __func__ << "(): num_samples is " << num_samples << "\n"; 
         std::cout << __func__ << "(): samples D16 Check:\n"; 
         all_close = cufftComplexes_are_close( samples_d16.data(), 
            exp_samples_d16, num_samples, max_diff, "samples_d16: ", debug);
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between actual samples_d16 from GPU and expected samples_d16."} };
         }
         std::cout << "\n"; 

         std::cout << __func__ << "(): conj_sqrs Check:\n"; 
         all_close = cufftComplexes_are_close( conj_sqrs.data(), 
            exp_conj_sqrs, num_samples, max_diff, "conj_sqrs: ", debug);
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between actual conj_sqrs from GPU and expected conj_sqrs."} };
         }
         std::cout << "\n"; 

         std::cout << __func__ << "(): conj_sqr_means Check:\n"; 
         all_close = cufftComplexes_are_close( conj_sqr_means.data(), 
            exp_conj_sqr_means, num_samples, max_diff, "conj_sqr_means: ", debug);
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between actual conj_sqr_means from GPU and expected conj_sqr_means."} };
         }
         std::cout << "\n"; 
      
         std::cout << __func__ << "(): conj_sqr_mean_mags Check:\n"; 
         all_close = vals_are_close( conj_sqr_mean_mags.data(), 
            exp_conj_sqr_mean_mags, num_samples, max_diff, "conj_sqr_mean_mags: ", debug);
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between actual conj_sqr_mean_mags from GPU and expected conj_sqr_mean_mags."} };
         }
         std::cout << "\n"; 

         std::cout << __func__ << "(): mag_sqrs Check:\n"; 
         all_close = vals_are_close( mag_sqrs.data(), 
            exp_mag_sqrs, num_samples, max_diff, "mag_sqrs: ", debug);
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between actual mag_sqrs from GPU and expected mag_sqrs."} };
         }
         std::cout << "\n"; 

         std::cout << __func__ << "(): mag_sqr_means Check:\n"; 
         all_close = vals_are_close( mag_sqr_means.data(), 
            exp_mag_sqr_means, num_samples, max_diff, "mag_sqr_means: ", debug);
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between actual mag_sqr_means from GPU and expected mag_sqr_means."} };
         }
         std::cout << "\n"; 

         
         print_results( "Norms: " );
         std::cout << "\n"; 
      }
      dout << __func__ << "(): norms Check:\n"; 
      all_close = vals_are_close( norms.data(), exp_norms, num_samples, max_diff, "norms: ", debug );
      if (!all_close) {
         throw std::runtime_error{ std::string{__func__} + 
            std::string{"(): Mismatch between actual norms from GPU and expected norms."} };
      }
      dout << "\n"; 
      
      std::cout << "All " << num_samples << " Norm Values matched expected values. Test Passed.\n\n"; 
      std::cout << "It took the GPU " << gpu_milliseconds 
         << " milliseconds to process " << num_samples 
         << " samples\n";

      std::cout << "That's a rate of " << ( (num_samples*1000.f)/gpu_milliseconds ) << " samples processed per second\n"; 


   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): " << ex.what() << "\n"; 
   }
}


void NormAutocorrGPU::calc_norms() {
   
   for( int index = 0; index < num_samples; ++index ) {
      if ( exp_mag_sqr_means[index] > 0 ) {
         exp_norms[index] = exp_conj_sqr_mean_mags[index]/exp_mag_sqr_means[index];
      } else {
         exp_norms[index] = 0.f;
      }
   } 

}


void NormAutocorrGPU::calc_mags() {
   
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_mean_mags[index] = hipCabsf( exp_conj_sqr_means[index] );
   } 

}


void NormAutocorrGPU::calc_complex_mag_squares() {

   for( int index = 0; index < num_samples; ++index ) {
      float temp = hipCabsf( samples[index] );
      exp_mag_sqrs[index] = temp * temp;
   } 
}


void NormAutocorrGPU::calc_auto_corrs() {
   
   dout << __func__ << "() start\n";
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqrs[index] = hipCmulf( samples[index], hipConjf( exp_samples_d16[index] ) );
   } 
   dout << __func__ << "() end\n";
}

void NormAutocorrGPU::calc_exp_conj_sqr_means() {

   // exp_conj_sqr_means must already be all zeros
   debug_printf( debug, "%s(): exp_conj_sqr_means[0] = { %f, %f }\n", __func__, exp_conj_sqr_means[0].x, exp_conj_sqr_means[0].y ); 
   for( int index = 0; index < conj_sqrs_window_size; ++index ) {
      exp_conj_sqr_means[0] = hipCaddf( exp_conj_sqr_means[0], exp_conj_sqrs[index] );
   }
   debug_printf( debug, "%s(): after initial sum: exp_conj_sqr_means[0] = { %f, %f }\n", __func__, exp_conj_sqr_means[0].x, exp_conj_sqr_means[0].y ); 
      
   int num_sums = num_samples - conj_sqrs_window_size;
   debug_printf( debug, "%s(): num_sums is %d\n", __func__, num_sums ); 
   for( int index = 1; index < num_sums; ++index ) {
      hipfftComplex temp = hipCsubf( exp_conj_sqr_means[index-1], exp_conj_sqrs[index-1] );
      exp_conj_sqr_means[index] = hipCaddf( temp, exp_conj_sqrs[index + conj_sqrs_window_size-1] );
   } 

   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_means[index] = complex_divide_by_scalar( exp_conj_sqr_means[index], (float)conj_sqrs_window_size );
   } 
}


void NormAutocorrGPU::calc_exp_mag_sqr_means() {

   debug_printf( debug, "%s(): exp_mag_sqr_means[0] = %f\n", __func__, exp_mag_sqr_means[0] ); 
   // exp_mag_sqr_means must already be all zeros
   for( int index = 0; index < mag_sqrs_window_size; ++index ) {
      exp_mag_sqr_means[0] = exp_mag_sqr_means[0] + exp_mag_sqrs[index];
   }
   debug_printf( debug, "%s(): After initial sum: exp_mag_sqr_means[0] = %f\n", __func__, exp_mag_sqr_means[0] ); 
      
   int num_sums = num_samples - mag_sqrs_window_size;
   for( int index = 1; index < num_sums; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index-1] - exp_mag_sqrs[index-1] + exp_mag_sqrs[index + mag_sqrs_window_size-1];
   } 

   for( int index = 0; index  < num_samples; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index]/(float)mag_sqrs_window_size;
   } 
}


void NormAutocorrGPU::gen_expected_norms() {
    
   dout << "num_samples is " << num_samples << "\n";

   float cpu_milliseconds = 0.f;
   Time_Point start = Steady_Clock::now();

   delay_vals16();
   calc_auto_corrs();
   calc_exp_conj_sqr_means();
   calc_mags();
   
   calc_complex_mag_squares();
   calc_exp_mag_sqr_means();
   
   calc_norms();

   Duration_ms duration_ms = Steady_Clock::now() - start;
   cpu_milliseconds = duration_ms.count();

   std::cout << "It took the CPU " << cpu_milliseconds << " milliseconds to process " << num_samples << " samples\n";
   std::cout << "That's a rate of " << ((num_samples*1000.f)/cpu_milliseconds) << " samples processed per second\n\n"; 
}
