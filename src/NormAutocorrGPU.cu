#include <hip/hip_runtime.h>

#include "my_utils.hpp"
#include "my_cuda_utils.hpp"
#include "my_cufft_utils.hpp"

#include "norm_autocorr_kernel.cuh"


#include "device_allocator.hpp"
#include "managed_allocator_host.hpp"
#include "managed_allocator_global.hpp"

#include "NormAutocorrGPU.cuh"

void NormAutocorrGPU::run() {
   try {
      hipError_t cerror = hipSuccess;
      int num_shared_bytes = 0;
      int threads_per_block = 64;
      int num_blocks = (num_samples + threads_per_block - 1) / threads_per_block;

      debug_cout( debug, __func__, "(): num_samples is ", num_samples, "\n" ); 
      debug_cout( debug, __func__, "(): threads_per_block is ", threads_per_block, "\n" ); 
      debug_cout( debug, __func__, "(): num_blocks is ", num_blocks, "\n" ); 

      gen_data();
      
      debug_cout( debug, __func__, "(): samples.size() is ", samples.size(), "\n" ); 
      
      print_cufftComplexes( samples.data(), num_samples, "Samples: ", " ", "\n" ); 

      hipStreamAttachMemAsync( *(stream_ptr.get()), samples.data(), 0, hipMemAttachGlobal );

      norm_autocorr_kernel<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
         norms.data(), 
         mag_sqr_means.data(), 
         mag_sqrs.data(), 
         conj_sqr_mean_mags.data(), 
         conj_sqr_means.data(), 
         conj_sqrs.data(), 
         samples_d16.data(), 
         samples.data(),
         conj_window_size,
         mag_sqrs_window_size,
         num_samples 
      );

      // Prefetch fspecs from the GPU
      hipStreamAttachMemAsync( *(stream_ptr.get()), norms.data(), 0, hipMemAttachHost );   
      
      try_cuda_func_throw( cerror, hipStreamSynchronize( *(stream_ptr.get())  ) );
      
      // norms.size() is 0 because the add_kernel modified the data and not a std::vector function
      debug_cout( debug, __func__, "(): norms.size() is ", norms.size(), "\n" ); 

      print_results( "Norms: " );
      std::cout << "\n"; 

   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): " << ex.what() << "\n"; 
   }
}


