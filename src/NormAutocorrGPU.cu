#include <hip/hip_runtime.h>

#include "my_utils.hpp"
#include "my_cuda_utils.hpp"
#include "my_cufft_utils.hpp"

#include "device_allocator.hpp"
#include "managed_allocator_host.hpp"
#include "managed_allocator_global.hpp"

#include "NormAutocorrGPU.cuh"

#include "norm_autocorr_kernel.cuh"

void NormAutocorrGPU::run() {
   try {
      hipError_t cerror = hipSuccess;
      int num_shared_bytes = 0;
      int threads_per_block = 1024;
      int num_blocks = (num_samples + threads_per_block - 1) / threads_per_block;

      debug_cout( debug, __func__, "(): num_samples is ", num_samples, "\n" ); 
      debug_cout( debug, __func__, "(): threads_per_block is ", threads_per_block, "\n" ); 
      debug_cout( debug, __func__, "(): num_blocks is ", num_blocks, "\n" ); 

      gen_data();
      gen_expected_norms();
      
      debug_cout( debug, __func__, "(): num_samples is ", num_samples, "\n" ); 
      
      if ( debug ) {
         print_cufftComplexes( samples.data(), num_samples, "Samples: ", " ", "\n" ); 
         print_cufftComplexes( exp_samples_d16, num_samples, "Expected Samples D16: ", " ", "\n" ); 
         //print_cufftComplexes( exp_conj_sqrs, num_samples, "Expected Conjugate Squares: ", " ", "\n" );
         //print_cufftComplexes( exp_conj_sqr_means, num_samples, "Expected Conjugate Square Means: ", " ", "\n" );
         //print_vals( exp_conj_sqr_mean_mags, num_samples, "Expected Conjugate Square Mean Mags: ", " ", "\n" ); 
         //print_vals( exp_mag_sqrs, num_samples, "Expected Magnitude Squares: ", " ", "\n" ); 
         //print_vals( exp_mag_sqr_means, num_samples, "Expected Magnitude Square Means: ", " ", "\n" );
         //print_vals( exp_norms, num_samples, "Expected Norms: ", " ", "\n" ); 
      }
      hipStreamAttachMemAsync( *(stream_ptr.get()), samples.data(), 0, hipMemAttachGlobal );

      norm_autocorr_kernel<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
         norms.data(), 
         mag_sqr_means.data(), 
         mag_sqrs.data(), 
         conj_sqr_mean_mags.data(), 
         conj_sqr_means.data(), 
         conj_sqrs.data(), 
         samples_d16.data(), 
         samples.data(),
         conj_sqrs_window_size,
         mag_sqrs_window_size,
         num_samples 
      );

      //// Prefetch fspecs from the GPU
      hipStreamAttachMemAsync( *(stream_ptr.get()), norms.data(), 0, hipMemAttachHost );   
      
      try_cuda_func_throw( cerror, hipStreamSynchronize( *(stream_ptr.get())  ) );
      
      // num_samples is 0 because the add_kernel modified the data and not a std::vector function
      debug_cout( debug, __func__, "(): num_samples is ", num_samples, "\n" ); 

      //print_results( "Norms: " );
      std::cout << "\n"; 

      float max_diff = 1e-1;
      bool all_close = cufftComplexes_are_close( samples_d16.data(), 
         exp_samples_d16, num_samples, max_diff, true);
      if (!all_close) {
         throw std::runtime_error{ std::string{__func__} + 
            std::string{"(): Mismatch between actual samples_d16 from GPU and expected samples_d16."} };
      }

      //bool all_close = vals_are_close( norms.data(), exp_norms, num_samples, max_diff, debug );
      //if (!all_close) {
      //   throw std::runtime_error{ std::string{__func__} + 
      //      std::string{"(): Mismatch between actual norms from GPU and expected norms."} };
      //}

   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): " << ex.what() << "\n"; 
   }
}


void NormAutocorrGPU::calc_norms() {
   
   for( int index = 0; index < num_samples; ++index ) {
      if ( exp_mag_sqr_means[index] > 0 ) {
         exp_norms[index] = exp_conj_sqr_mean_mags[index]/exp_mag_sqr_means[index];
      } else {
         exp_norms[index] = 0.f;
      }
   } 

}


void NormAutocorrGPU::calc_mags() {
   
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_mean_mags[index] = hipCabsf( exp_conj_sqr_means[index] );
   } 

}


void NormAutocorrGPU::calc_complex_mag_squares() {

   for( int index = 0; index < num_samples; ++index ) {
      float temp = hipCabsf( samples[index] );
      exp_mag_sqrs[index] = temp * temp;
   } 
}


void NormAutocorrGPU::calc_auto_corrs() {
   
   dout << __func__ << "() start\n";
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqrs[index] = hipCmulf( samples[index], hipConjf( exp_samples_d16[index] ) );
   } 
   dout << __func__ << "() end\n";
}

void NormAutocorrGPU::calc_exp_conj_sqr_means() {

   // exp_conj_sqr_means must already be all zeros
   for( int index = 0; index < conj_sqrs_window_size; ++index ) {
      exp_conj_sqr_means[0] = hipCaddf( exp_conj_sqr_means[0], exp_conj_sqrs[index] );
   }
      
   int num_sums = num_samples - conj_sqrs_window_size;
   for( int index = 1; index < num_sums; ++index ) {
      exp_conj_sqr_means[index] = hipCsubf( hipCaddf( exp_conj_sqr_means[index-1], exp_conj_sqrs[index + conj_sqrs_window_size-1] ), 
         exp_conj_sqrs[index-1] );
   } 

   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_means[index] = complex_divide_by_scalar( exp_conj_sqr_means[index], (float)conj_sqrs_window_size );
   } 
}


void NormAutocorrGPU::calc_exp_mag_sqr_means() {

   // exp_mag_sqr_means must already be all zeros
   for( int index = 0; index < mag_sqrs_window_size; ++index ) {
      exp_mag_sqr_means[0] = exp_mag_sqr_means[0] + exp_mag_sqrs[index];
   }
      
   int num_sums = num_samples - mag_sqrs_window_size;
   for( int index = 1; index < num_sums; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index-1] + exp_mag_sqrs[index + mag_sqrs_window_size-1] - exp_mag_sqrs[index-1];
   } 

   for( int index = 0; index  < num_samples; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index]/(float)mag_sqrs_window_size;
   } 
}


void NormAutocorrGPU::gen_expected_norms() {
    
   dout << "num_samples is " << num_samples << "\n";

   delay_vals16();
   calc_auto_corrs();
   calc_exp_conj_sqr_means();
   calc_mags();
   
   calc_complex_mag_squares();
   calc_exp_mag_sqr_means();
   
   calc_norms();
}
