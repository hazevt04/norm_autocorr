#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "my_utils.hpp"
#include "my_cuda_utils.hpp"
#include "my_cufft_utils.hpp"

#include "NormAutocorrGPU.hpp"

#include "norm_autocorr_kernel.cuh"

void NormAutocorrGPU::initialize_samples( const int seed = 0, const bool debug = false ) {
   try {
      samples.resize(adjusted_num_samples);
      std::fill( samples.begin(), samples.end(), make_hipFloatComplex(0.f,0.f) );

      if( test_select_string =="Sinusoidal" ) {
         dout << __func__ << "(): Sinusoidal Sample Test Selected\n";
         for( size_t index = 0; index < num_samples; ++index ) {
            float t_val_real = AMPLITUDE*sin(2*PI*FREQ*index);
            float t_val_imag = AMPLITUDE*sin(2*PI*FREQ*index);
            samples[index] = make_hipFloatComplex( t_val_real, t_val_imag );
         } 
      } else if ( test_select_string == "Random" ) {
         dout << __func__ << "(): Random Sample Test Selected\n";
         gen_cufftComplexes( samples.data(), num_samples, -50.0, 50.0 );
      } else if ( test_select_string == "Filebased" ) {
         dout << __func__ << "(): File-Based Sample Test Selected. File is " << filename << "\n";
         read_binary_file<hipfftComplex>( 
            samples,
            filepath.c_str(),
            num_samples, 
            debug );
      } else {
         throw std::runtime_error( std::string{__func__} + 
            std::string{"(): Error: Invalid test select: "} + 
               test_select_string );
      }            
      if (debug) {
         print_cufftComplexes( samples.data(), num_samples, "Samples: ",  " ",  "\n" ); 
      }
   } catch( std::exception& ex ) {
      throw std::runtime_error{
         std::string{__func__} + std::string{"(): "} + ex.what()
      }; 
   } // end of try
} // end of initialize_samples( const NormAutocorrGPU::TestSelect_e test_select = Sinusoidal, 


NormAutocorrGPU::NormAutocorrGPU( 
   const my_args_t& args
):
      num_samples( args.num_samples ),
      conj_sqrs_window_size( args.conj_sqrs_window_size ),
      mag_sqrs_window_size( args.mag_sqrs_window_size ),
      max_num_iters( args.max_num_iters ),
      test_select_string( args.test_select_string ),
      filename( args.filename ),
      exp_norms_filename( args.exp_norms_filename ),
      debug( args.debug ) {

   try {
      hipError_t cerror = hipSuccess;         
      try_cuda_func_throw( cerror, hipGetDevice( &device_id ) );

      stream_ptr = my_make_unique<hipStream_t>();
      try_cudaStreamCreate( stream_ptr.get() );
      dout << __func__ << "(): after hipStreamCreate()\n"; 

      dout << __func__ << "(): num_samples is " << num_samples << "\n";

      num_blocks = (num_samples + (threads_per_block-1))/threads_per_block;
      dout << __func__ << "(): num_blocks is " << num_blocks << "\n";

      adjusted_num_samples = threads_per_block * num_blocks;
      adjusted_num_sample_bytes = adjusted_num_samples * sizeof( hipfftComplex );
      adjusted_num_norm_bytes = adjusted_num_samples * sizeof( float );
      num_norm_bytes = adjusted_num_samples * sizeof( float );

      dout << __func__ << "(): adjusted number of samples for allocation is " 
         << adjusted_num_samples << "\n";
      dout << __func__ << "(): adjusted number of sample bytes for hipMemcpyAsync is "
         << adjusted_num_sample_bytes << "\n";
      dout << __func__ << "(): adjusted number of norm bytes for hipMemcpyAsync is "
         << adjusted_num_norm_bytes << "\n\n";

      samples.reserve( adjusted_num_samples );
      
      //d_samples.reserve( adjusted_num_samples );
      samples_d16.reserve( adjusted_num_samples );
      conj_sqrs.reserve( adjusted_num_samples );
      conj_sqr_means.reserve( adjusted_num_samples );
      conj_sqr_mean_mags.reserve( adjusted_num_samples );
      mag_sqrs.reserve( adjusted_num_samples );
      mag_sqr_means.reserve( adjusted_num_samples );
      norms.reserve( adjusted_num_samples );
      //d_norms.reserve( adjusted_num_samples );

      samples.resize(adjusted_num_samples); 
      norms.resize(adjusted_num_samples);
      std::fill( norms.begin(), norms.end(), 0 );
      
      try_cuda_func_throw( cerror, hipHostGetDevicePointer( &d_samples, samples.data(), 0 ) );
      try_cuda_func_throw( cerror, hipHostGetDevicePointer( &d_norms, norms.data(), 0 ) );

      //try_cuda_func_throw( cerror, hipMemset( d_samples.data(), adjusted_num_sample_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( samples_d16.data(), adjusted_num_sample_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( conj_sqrs.data(), adjusted_num_sample_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( conj_sqr_means.data(), adjusted_num_sample_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( conj_sqr_mean_mags.data(), adjusted_num_sample_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( mag_sqrs.data(), adjusted_num_norm_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( mag_sqr_means.data(), adjusted_num_norm_bytes, 0 ) );
      //try_cuda_func_throw( cerror, hipMemset( d_norms.data(), adjusted_num_norm_bytes, 0 ) );
      
      exp_samples_d16.resize(num_samples);
      exp_conj_sqrs.resize(num_samples);
      exp_conj_sqr_means.resize(num_samples);
      exp_conj_sqr_mean_mags.resize(num_samples);
      exp_mag_sqrs.resize(num_samples);
      exp_mag_sqr_means.resize(num_samples);
      exp_norms.resize(num_samples);

      exp_samples_d16.reserve(num_samples);
      exp_conj_sqrs.reserve(num_samples);
      exp_conj_sqr_means.reserve(num_samples);
      exp_conj_sqr_mean_mags.reserve(num_samples);
      exp_mag_sqrs.reserve(num_samples);
      exp_mag_sqr_means.reserve(num_samples);
      exp_norms.reserve(num_samples);
      
      for( int index = 0; index < num_samples; ++index ) {
         exp_samples_d16[index] = make_hipFloatComplex(0.f,0.f);
         exp_conj_sqrs[index] =  make_hipFloatComplex(0.f,0.f);
         exp_conj_sqr_means[index] = make_hipFloatComplex(0.f,0.f);
         exp_mag_sqrs[index] = 0.f;
         exp_mag_sqr_means[index] = 0.f;
         exp_norms[index] = 0.f;
      } 

      initialize_samples();

      char* user_env = getenv( "USER" );
      if ( user_env == nullptr ) {
         throw std::runtime_error( std::string{__func__} + 
            "(): Empty USER env. USER environment variable needed for paths to files" ); 
      }
      
      std::string filepath_prefix = "/home/" + std::string{user_env} + "/Sandbox/CUDA/norm_autocorr/";

      filepath = filepath_prefix + filename;
      exp_norms_filepath = filepath_prefix + exp_norms_filename;

      dout << "Filepath is " << filepath << "\n";
      dout << "Expected Norms Filepath is " << exp_norms_filepath << "\n";

   } catch( std::exception& ex ) {
      throw std::runtime_error{
         std::string{__func__} + std::string{"(): "} + ex.what()
      }; 
   }
} // end of constructor


void NormAutocorrGPU::run() {
   try {
      hipError_t cerror = hipSuccess;
      int num_shared_bytes = 0;

      dout << __func__ << "(): num_samples is " << num_samples << "\n"; 
      dout << __func__ << "(): threads_per_block is " << threads_per_block << "\n"; 
      dout << __func__ << "(): num_blocks is " << num_blocks << "\n\n"; 
      
      dout << __func__ << "(): adjusted_num_samples is " << adjusted_num_samples << "\n"; 
      dout << __func__ << "(): adjusted_num_sample_bytes is " << adjusted_num_sample_bytes << "\n"; 
      dout << __func__ << "(): adjusted_num_norm_bytes is " << adjusted_num_norm_bytes << "\n"; 

      gen_expected_norms();

      if ( debug ) {
         print_cufftComplexes( exp_samples_d16.data(), num_samples, "Expected Samples D16: ", " ", "\n" ); 
         print_cufftComplexes( exp_conj_sqrs.data(), num_samples, "Expected Conjugate Squares: ", " ", "\n" );
         print_cufftComplexes( exp_conj_sqr_means.data(), num_samples, "Expected Conjugate Square Means: ", " ", "\n" );
         print_vals( exp_conj_sqr_mean_mags.data(), num_samples, "Expected Conjugate Square Mean Mags: ", " ", "\n" ); 
         print_vals( exp_mag_sqrs.data(), num_samples, "Expected Magnitude Squares: ", " ", "\n" ); 
         print_vals( exp_mag_sqr_means.data(), num_samples, "Expected Magnitude Square Means: ", " ", "\n" );
         print_vals( exp_norms.data(), num_samples, "Expected Norms: ", " ", "\n" ); 
      }
      
      float gpu_milliseconds = 0.f;
      Time_Point start = Steady_Clock::now();
      
      //try_cuda_func( cerror, hipMemcpyAsync( d_samples.data(), samples.data(), adjusted_num_sample_bytes,
      //         hipMemcpyHostToDevice, *(stream_ptr.get()) ) );
      //try_cuda_func_throw( cerror, hipMemPrefetchAsync( d_samples, adjusted_num_sample_bytes, 
      //   device_id, *(stream_ptr.get()) ) );

      dout << __func__ << "(): Running norm_autocorr_kernel...\n";
      norm_autocorr_kernel<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
         d_norms, 
         mag_sqr_means.data(), 
         mag_sqrs.data(), 
         conj_sqr_mean_mags.data(), 
         conj_sqr_means.data(), 
         conj_sqrs.data(), 
         samples_d16.data(), 
         d_samples,
         conj_sqrs_window_size,
         mag_sqrs_window_size,
         num_samples 
      );

      //try_cuda_func( cerror, hipMemcpyAsync( norms.data(), d_norms.data(), adjusted_num_norm_bytes,
      //         hipMemcpyDeviceToHost, *(stream_ptr.get()) ) );
      //try_cuda_func_throw( cerror, hipMemPrefetchAsync( d_norms, adjusted_num_norm_bytes, 
      //   device_id, *(stream_ptr.get()) ) );     
      
      try_cuda_func_throw( cerror, hipDeviceSynchronize() );
      
      Duration_ms duration_ms = Steady_Clock::now() - start;
      gpu_milliseconds = duration_ms.count();
      dout << __func__ << "(): norm_autocorr_kernel took " << gpu_milliseconds << " ms\n";

      float max_diff = 1;
      bool all_close = false;
      if ( debug ) {
         print_results( "Norms: " );
         std::cout << "\n"; 
      }
      dout << __func__ << "(): norms Check:\n"; 
      all_close = vals_are_close( norms.data(), exp_norms.data(), num_samples, max_diff, "norms: ", debug );
      if (!all_close) {
         throw std::runtime_error{ std::string{__func__} + 
            std::string{"(): Mismatch between actual norms from GPU and expected norms."} };
      }
      dout << "\n"; 
      
      std::cout << "All " << num_samples << " Norm Values matched expected values. Test Passed.\n\n"; 
      std::cout << "It took the GPU " << gpu_milliseconds 
         << " milliseconds to process " << num_samples 
         << " samples\n";

      std::cout << "That's a rate of " << ( (num_samples*1000.f)/gpu_milliseconds ) << " samples processed per second for the GPU\n"; 


   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): " << ex.what() << "\n"; 
   }
}


void NormAutocorrGPU::calc_norms() {
   
   for( int index = 0; index < num_samples; ++index ) {
      if ( exp_mag_sqr_means[index] > 0 ) {
         exp_norms[index] = exp_conj_sqr_mean_mags[index]/exp_mag_sqr_means[index];
      } else {
         exp_norms[index] = 0.f;
      }
   } 

}


void NormAutocorrGPU::calc_mags() {
   
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_mean_mags[index] = hipCabsf( exp_conj_sqr_means[index] );
   } 

}


void NormAutocorrGPU::calc_complex_mag_squares() {

   for( int index = 0; index < num_samples; ++index ) {
      float temp = hipCabsf( samples[index] );
      exp_mag_sqrs[index] = temp * temp;
   } 
}


void NormAutocorrGPU::calc_auto_corrs() {
   
   dout << __func__ << "() start\n";
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqrs[index] = hipCmulf( samples[index], hipConjf( exp_samples_d16[index] ) );
   } 
   dout << __func__ << "() end\n";
}

void NormAutocorrGPU::calc_exp_conj_sqr_means() {

   // exp_conj_sqr_means must already be all zeros
   dout << __func__ << "(): exp_conj_sqr_means[0] = { " 
      << exp_conj_sqr_means[0].x << ", " << exp_conj_sqr_means[0].y << " }\n"; 
   for( int index = 0; index < conj_sqrs_window_size; ++index ) {
      exp_conj_sqr_means[0] = hipCaddf( exp_conj_sqr_means[0], exp_conj_sqrs[index] );
   }
   dout << __func__ << "(): after initial summation, exp_conj_sqr_means[0] = { " 
      << exp_conj_sqr_means[0].x << ", " << exp_conj_sqr_means[0].y << " }\n"; 
      
   int num_sums = num_samples - conj_sqrs_window_size;
   dout << __func__ << "(): num_sums is " << num_sums << "\n"; 
   for( int index = 1; index < num_sums; ++index ) {
      hipfftComplex temp = hipCsubf( exp_conj_sqr_means[index-1], exp_conj_sqrs[index-1] );
      exp_conj_sqr_means[index] = hipCaddf( temp, exp_conj_sqrs[index + conj_sqrs_window_size-1] );
   } 

   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_means[index] = complex_divide_by_scalar( exp_conj_sqr_means[index], (float)conj_sqrs_window_size );
   } 
}


void NormAutocorrGPU::calc_exp_mag_sqr_means() {

   dout << __func__ << "(): exp_mag_sqr_means[0] = " << exp_mag_sqr_means[0] << "\n"; 
   // exp_mag_sqr_means must already be all zeros
   for( int index = 0; index < mag_sqrs_window_size; ++index ) {
      exp_mag_sqr_means[0] = exp_mag_sqr_means[0] + exp_mag_sqrs[index];
   }
   dout << __func__ << "(): After initial sum, exp_mag_sqr_means[0] = " << exp_mag_sqr_means[0] << "\n"; 
    
   int num_sums = num_samples - mag_sqrs_window_size;
   for( int index = 1; index < num_sums; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index-1] - exp_mag_sqrs[index-1] + exp_mag_sqrs[index + mag_sqrs_window_size-1];
   } 

   for( int index = 0; index  < num_samples; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index]/(float)mag_sqrs_window_size;
   } 
}


void NormAutocorrGPU::cpu_run() {
   try { 
      float cpu_milliseconds = 0.f;
      
      dout << __func__ << "(): num_samples is " << num_samples << "\n";
      
      Time_Point start = Steady_Clock::now();

      delay_vals16();
      calc_auto_corrs();
      calc_exp_conj_sqr_means();
      calc_mags();
      
      calc_complex_mag_squares();
      calc_exp_mag_sqr_means();
      
      calc_norms();

      Duration_ms duration_ms = Steady_Clock::now() - start;
      cpu_milliseconds = duration_ms.count();

      std::cout << "It took the CPU " << cpu_milliseconds << " milliseconds to process " << num_samples << " samples\n";
      std::cout << "That's a rate of " << ((num_samples*1000.f)/cpu_milliseconds) << " samples processed per second\n\n"; 

   } catch( std::exception& ex ) {
      throw std::runtime_error( std::string{__func__} +  std::string{"(): "} + ex.what() ); 
   }
}


void NormAutocorrGPU::gen_expected_norms() {
   try { 

      cpu_run();

      if ( test_select_string == "Filebased" ) {
         float norms_from_file[num_samples];
         read_binary_file<float>( norms_from_file, exp_norms_filepath.c_str(), num_samples, debug );

         float max_diff = 1.f;
         bool all_close = false;
         dout << __func__ << "(): Exp Norms Check Against File:\n"; 
         all_close = vals_are_close( exp_norms.data(), norms_from_file, num_samples, max_diff, "exp norms: ", debug );
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between expected norms and norms from file."} };
         }
         dout << "\n";
      }

   } catch( std::exception& ex ) {
      throw std::runtime_error( std::string{__func__} +  std::string{"(): "} + ex.what() ); 
   }

}

// private function
void NormAutocorrGPU::delay_vals16() {
   
   dout << __func__ << "() start\n";
   dout << __func__ << "() samples.size() is " << samples.size() << "\n";
   dout << __func__ << "() samples[0] is " << samples[0] << "\n";
   dout << __func__ << "() samples[1] is " << samples[1] << "\n";

   for( int index = 0; index < num_samples; ++index ) {
      if ( index < 16 ) {
         exp_samples_d16[index] = make_hipFloatComplex(0.f, 0.f);
      } else {
         exp_samples_d16[index] = samples[index-16]; 
      }
   } 

   dout << __func__ << "() exp_samples_d16[15] is " << exp_samples_d16[15] << "\n";
   dout << __func__ << "() exp_samples_d16[16] is " << exp_samples_d16[16] << "\n";
   dout << __func__ << "() exp_samples_d16[17] is " << exp_samples_d16[17] << "\n";
   dout << __func__ << "() done\n";
} // end of delay_vals16 

NormAutocorrGPU::~NormAutocorrGPU() {
   dout << "dtor called\n";
   //d_samples.clear();    
   samples.clear();    
   samples_d16.clear();
   conj_sqrs.clear();
   conj_sqr_means.clear();
   conj_sqr_mean_mags.clear();
   mag_sqrs.clear();
   mag_sqr_means.clear();
   norms.clear();
   //d_norms.clear();

   /*delete [] exp_samples_d16;*/
   /*if ( exp_conj_sqrs ) delete [] exp_conj_sqrs;*/
   /*if ( exp_conj_sqr_means ) delete [] exp_conj_sqr_means;*/
   /*if ( exp_conj_sqr_mean_mags ) delete [] exp_conj_sqr_mean_mags;*/
   /*if ( exp_mag_sqrs ) delete [] exp_mag_sqrs;*/
   /*if ( exp_mag_sqr_means ) delete [] exp_mag_sqr_means;*/
   /*if ( exp_norms ) delete [] exp_norms;*/
   exp_samples_d16.clear();
   exp_conj_sqrs.clear();
   exp_conj_sqr_means.clear();
   exp_conj_sqr_mean_mags.clear();
   exp_mag_sqrs.clear();
   exp_mag_sqr_means.clear();
   exp_norms.clear();

   if ( stream_ptr ) hipStreamDestroy( *(stream_ptr.get()) );
   
   dout << "dtor done\n";
} // end of destructor

