#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "my_utils.hpp"
#include "my_cuda_utils.hpp"
#include "my_cufft_utils.hpp"

#include "NormAutocorrGPU.cuh"

#include "norm_autocorr_kernel.cuh"

void NormAutocorrGPU::run() {
   try {
      hipError_t cerror = hipSuccess;
      int num_shared_bytes = 0;

      dout << __func__ << "(): num_samples is " << num_samples << "\n"; 
      dout << __func__ << "(): threads_per_block is " << threads_per_block << "\n"; 
      dout << __func__ << "(): num_blocks is " << num_blocks << "\n\n"; 
      
      dout << __func__ << "(): adjusted_num_samples is " << adjusted_num_samples << "\n"; 
      dout << __func__ << "(): adjusted_num_sample_bytes is " << adjusted_num_sample_bytes << "\n"; 
      dout << __func__ << "(): adjusted_num_norm_bytes is " << adjusted_num_norm_bytes << "\n"; 

      gen_expected_norms();

      if ( debug ) {
         print_cufftComplexes( exp_samples_d16, num_samples, "Expected Samples D16: ", " ", "\n" ); 
         print_cufftComplexes( exp_conj_sqrs, num_samples, "Expected Conjugate Squares: ", " ", "\n" );
         print_cufftComplexes( exp_conj_sqr_means, num_samples, "Expected Conjugate Square Means: ", " ", "\n" );
         print_vals( exp_conj_sqr_mean_mags, num_samples, "Expected Conjugate Square Mean Mags: ", " ", "\n" ); 
         print_vals( exp_mag_sqrs, num_samples, "Expected Magnitude Squares: ", " ", "\n" ); 
         print_vals( exp_mag_sqr_means, num_samples, "Expected Magnitude Square Means: ", " ", "\n" );
         print_vals( exp_norms, num_samples, "Expected Norms: ", " ", "\n" ); 
      }
      
      float gpu_milliseconds = 0.f;
      Time_Point start = Steady_Clock::now();
      
      try_cuda_func( cerror, hipMemcpyAsync( d_samples.data(), samples.data(), adjusted_num_sample_bytes,
               hipMemcpyHostToDevice, *(stream_ptr.get()) ) );

      norm_autocorr_kernel<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
         d_norms.data(), 
         mag_sqr_means.data(), 
         mag_sqrs.data(), 
         conj_sqr_mean_mags.data(), 
         conj_sqr_means.data(), 
         conj_sqrs.data(), 
         samples_d16.data(), 
         d_samples.data(),
         conj_sqrs_window_size,
         mag_sqrs_window_size,
         num_samples 
      );

      try_cuda_func( cerror, hipMemcpyAsync( norms.data(), d_norms.data(), adjusted_num_norm_bytes,
               hipMemcpyDeviceToHost, *(stream_ptr.get()) ) );
      
      try_cuda_func_throw( cerror, hipDeviceSynchronize() );
      
      Duration_ms duration_ms = Steady_Clock::now() - start;
      gpu_milliseconds = duration_ms.count();

      float max_diff = 1;
      bool all_close = false;
      if ( debug ) {
         print_results( "Norms: " );
         std::cout << "\n"; 
      }
      dout << __func__ << "(): norms Check:\n"; 
      all_close = vals_are_close( norms.data(), exp_norms, num_samples, max_diff, "norms: ", debug );
      if (!all_close) {
         throw std::runtime_error{ std::string{__func__} + 
            std::string{"(): Mismatch between actual norms from GPU and expected norms."} };
      }
      dout << "\n"; 
      
      std::cout << "All " << num_samples << " Norm Values matched expected values. Test Passed.\n\n"; 
      std::cout << "It took the GPU " << gpu_milliseconds 
         << " milliseconds to process " << num_samples 
         << " samples\n";

      std::cout << "That's a rate of " << ( (num_samples*1000.f)/gpu_milliseconds ) << " samples processed per second\n"; 


   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): " << ex.what() << "\n"; 
   }
}


void NormAutocorrGPU::calc_norms() {
   
   for( int index = 0; index < num_samples; ++index ) {
      if ( exp_mag_sqr_means[index] > 0 ) {
         exp_norms[index] = exp_conj_sqr_mean_mags[index]/exp_mag_sqr_means[index];
      } else {
         exp_norms[index] = 0.f;
      }
   } 

}


void NormAutocorrGPU::calc_mags() {
   
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqr_mean_mags[index] = hipCabsf( exp_conj_sqr_means[index] );
   } 

}


void NormAutocorrGPU::calc_complex_mag_squares() {

   for( int index = 0; index < num_samples; ++index ) {
      float temp = hipCabsf( samples[index] );
      exp_mag_sqrs[index] = temp * temp;
   } 
}


void NormAutocorrGPU::calc_auto_corrs() {
   
   dout << __func__ << "() start\n";
   for( int index = 0; index < num_samples; ++index ) {
      exp_conj_sqrs[index] = hipCmulf( samples[index], hipConjf( exp_samples_d16[index] ) );
   } 
   dout << __func__ << "() end\n";
}

void NormAutocorrGPU::calc_exp_conj_sqr_means() {

   // exp_conj_sqr_means must already be all zeros
   dout << __func__ << "(): exp_conj_sqr_means[0] = { " 
      << exp_conj_sqr_means[0].x << ", " << exp_conj_sqr_means[0].y << " }\n"; 
   for( int index = 0; index < conj_sqrs_window_size; ++index ) {
      exp_conj_sqr_means[0] = hipCaddf( exp_conj_sqr_means[0], exp_conj_sqrs[index] );
   }
   dout << __func__ << "(): after initial summation, exp_conj_sqr_means[0] = { " 
      << exp_conj_sqr_means[0].x << ", " << exp_conj_sqr_means[0].y << " }\n"; 
      
   int num_sums = num_samples - conj_sqrs_window_size;
   dout << __func__ << "(): num_sums is " << num_sums << "\n"; 
   for( int index = 1; index < num_sums; ++index ) {
      hipfftComplex temp = hipCsubf( exp_conj_sqr_means[index-1], exp_conj_sqrs[index-1] );
      exp_conj_sqr_means[index] = hipCaddf( temp, exp_conj_sqrs[index + conj_sqrs_window_size-1] );
   } 

   /*for( int index = 0; index < num_samples; ++index ) {*/
   /*   exp_conj_sqr_means[index] = complex_divide_by_scalar( exp_conj_sqr_means[index], (float)conj_sqrs_window_size );*/
   /*} */
}


void NormAutocorrGPU::calc_exp_mag_sqr_means() {

   dout << __func__ << "(): exp_mag_sqr_means[0] = " << exp_mag_sqr_means[0] << "\n"; 
   // exp_mag_sqr_means must already be all zeros
   for( int index = 0; index < mag_sqrs_window_size; ++index ) {
      exp_mag_sqr_means[0] = exp_mag_sqr_means[0] + exp_mag_sqrs[index];
   }
   dout << __func__ << "(): After initial sum, exp_mag_sqr_means[0] = " << exp_mag_sqr_means[0] << "\n"; 
    
   int num_sums = num_samples - mag_sqrs_window_size;
   for( int index = 1; index < num_sums; ++index ) {
      exp_mag_sqr_means[index] = exp_mag_sqr_means[index-1] - exp_mag_sqrs[index-1] + exp_mag_sqrs[index + mag_sqrs_window_size-1];
   } 

   /*for( int index = 0; index  < num_samples; ++index ) {*/
   /*   exp_mag_sqr_means[index] = exp_mag_sqr_means[index]/(float)mag_sqrs_window_size;*/
   /*} */
}


void NormAutocorrGPU::cpu_run() {
   try { 
      float cpu_milliseconds = 0.f;
      
      dout << __func__ << "(): num_samples is " << num_samples << "\n";
      
      Time_Point start = Steady_Clock::now();

      delay_vals16();
      calc_auto_corrs();
      calc_exp_conj_sqr_means();
      calc_mags();
      
      calc_complex_mag_squares();
      calc_exp_mag_sqr_means();
      
      calc_norms();

      Duration_ms duration_ms = Steady_Clock::now() - start;
      cpu_milliseconds = duration_ms.count();

      std::cout << "It took the CPU " << cpu_milliseconds << " milliseconds to process " << num_samples << " samples\n";
      std::cout << "That's a rate of " << ((num_samples*1000.f)/cpu_milliseconds) << " samples processed per second\n\n"; 

   } catch( std::exception& ex ) {
      throw std::runtime_error( std::string{__func__} +  std::string{"(): "} + ex.what() ); 
   }
}


void NormAutocorrGPU::gen_expected_norms() {
   try { 

      cpu_run();

      if ( test_select_string == "Filebased" ) {
         float norms_from_file[num_samples];
         read_binary_file<float>( norms_from_file, norm_filepath.c_str(), num_samples, debug );

         float max_diff = 1.f;
         bool all_close = false;
         dout << __func__ << "(): Exp Norms Check Against File:\n"; 
         all_close = vals_are_close( exp_norms, norms_from_file, num_samples, max_diff, "exp norms: ", debug );
         if (!all_close) {
            throw std::runtime_error{ std::string{__func__} + 
               std::string{"(): Mismatch between expected norms and norms from file."} };
         }
         dout << "\n";
      }

   } catch( std::exception& ex ) {
      throw std::runtime_error( std::string{__func__} +  std::string{"(): "} + ex.what() ); 
   }

}
