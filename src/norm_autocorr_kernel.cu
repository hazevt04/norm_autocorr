#include "hip/hip_runtime.h"

#include <stdio.h>

#include "my_cufft_utils.hpp"

#include "norm_autocorr_kernel.cuh"

__global__
void delay16( hipfftComplex* delayed_vals, const hipfftComplex* vals, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;

   if ( global_index < 16 ) {
      delayed_vals[global_index] = {0.f,0.f};
   } else if ( global_index < num_vals ) {
      delayed_vals[global_index] = vals[global_index-16];
   }

}

__global__
void auto_correlation( hipfftComplex* __restrict__ conj_sqrs, const hipfftComplex* __restrict__ samples_d16,
   const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      conj_sqrs[index] = hipCmulf( samples[index], hipConjf( samples_d16[index] ) );
   }
}


__global__
void calc_conj_sqr_sums( 
      hipfftComplex* __restrict__ conj_sqr_sums, 
      const hipfftComplex* __restrict__ conj_sqrs, 
      const int conj_sqr_window_size, 
      const int num_vals 
   ) { 

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      hipfftComplex  t_conj_sqr_sum = make_hipFloatComplex(0.0,0.0);

      for( int w_index = 0; w_index < conj_sqr_window_size; ++w_index ) {
         t_conj_sqr_sum = hipCaddf( t_conj_sqr_sum, conj_sqrs[index + w_index] );
      }
      //conj_sqr_sums[index] = complex_divide_by_scalar( t_conj_sqr_sum, (float)conj_sqr_window_size );
      conj_sqr_sums[index] = t_conj_sqr_sum;
   }

}

__global__
void calc_conj_sqr_sum_mags( float* __restrict__ conj_sqr_sum_mags, const hipfftComplex* __restrict__ conj_sqr_sums, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      conj_sqr_sum_mags[index] = hipCabsf( conj_sqr_sums[index] );
   }
}


__global__
void calc_mag_sqrs( float* __restrict__ mag_sqrs, const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      float temp = hipCabsf( samples[index] );
      mag_sqrs[index] = temp * temp;
   }
}


__global__
void calc_mag_sqr_sums( 
      float* __restrict__ mag_sqr_sums, 
      const float* __restrict__ mag_sqrs,
      const int mag_sqr_window_size, 
      const int num_vals 
   ) { 

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      float  t_mag_sqr_sum = 0.0;
      for( int w_index = 0; w_index < mag_sqr_window_size; ++w_index ) {
         t_mag_sqr_sum = t_mag_sqr_sum + mag_sqrs[index + w_index];
      }
      //mag_sqr_sums[index] = t_mag_sqr_sum/(float)mag_sqr_window_size;
      mag_sqr_sums[index] = t_mag_sqr_sum;
   }

}


__global__
void normalize( float* __restrict__ norms, const float* __restrict__ conj_sqr_sum_mags, 
   const float* __restrict__ mag_sqr_sums, const int num_samples ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_samples; index += stride) {
      if ( mag_sqr_sums[index] > 0.f ) {
         norms[index] =  conj_sqr_sum_mags[index]/mag_sqr_sums[index];
      } else {
         norms[index] = 0.f;
      }
   }

}

/*__global__*/
/*void norm_autocorr_kernel( */
/*   float* __restrict__ norms, */
/*   float* __restrict__ mag_sqr_sums, */
/*   float* __restrict__ mag_sqrs, */
/*   float* __restrict__ conj_sqr_sum_mags, */
/*   hipfftComplex* __restrict__ conj_sqr_sums, */
/*   hipfftComplex* __restrict__ conj_sqrs, */
/*   hipfftComplex* __restrict__ samples_d16, */
/*   const hipfftComplex* __restrict__ samples,*/
/*   const int conj_sqr_window_size, */
/*   const int mag_sqr_window_size,*/
/*   const int num_samples ) {*/

/*   int num_conj_sqr_sums = num_samples - conj_sqr_window_size;*/
/*   int num_mag_sqr_sums = num_samples - mag_sqr_window_size;*/

/*   delay16( samples_d16, samples, num_samples );*/
/*   __syncthreads();*/
/*   auto_correlation( conj_sqrs, samples_d16, samples, num_samples );*/
/*   __syncthreads();*/
   
/*   calc_conj_sqr_sums( */
/*      conj_sqr_sums, */
/*      conj_sqrs, */
/*      conj_sqr_window_size, */
/*      num_conj_sqr_sums );*/
/*   __syncthreads();*/

/*   calc_conj_sqr_sum_mags( conj_sqr_sum_mags, conj_sqr_sums, */
/*      num_conj_sqr_sums );*/
/*   __syncthreads();*/

/*   calc_mag_sqrs( mag_sqrs, samples, num_samples );*/
/*   __syncthreads();*/

/*   calc_mag_sqr_sums( */
/*      mag_sqr_sums, */
/*      mag_sqrs,*/
/*      mag_sqr_window_size, */
/*      num_mag_sqr_sums );*/
/*   __syncthreads();*/
   
/*   normalize( norms, conj_sqr_sum_mags, mag_sqr_sums, num_samples );*/
/*}*/
