#include "hip/hip_runtime.h"

#include <stdio.h>

#include "my_cufft_utils.hpp"

#include "norm_autocorr_kernel.cuh"

template<typename T>
__device__
void delay16( T* delayed_vals, const T* vals, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;

   if ( global_index < 16 ) {
      delayed_vals[global_index] = T{0};
   } else if ( global_index < num_vals ) {
      delayed_vals[global_index] = vals[global_index-16];
   }

}


template
__device__
void delay16<hipfftComplex>( hipfftComplex* delayed_vals, const hipfftComplex* vals, const int num_vals );


__device__
void auto_correlation( hipfftComplex* __restrict__ conj_sqrs, const hipfftComplex* __restrict__ samples_d16,
   const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      conj_sqrs[index] = hipCmulf( samples[index], hipConjf( samples_d16[index] ) );
   }
}


__device__
void auto_correlations( hipfftComplex* __restrict__ conj_sqrs, const hipfftComplex* __restrict__ samples_d16,
   const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int thread_index = threadIdx.x;

   printf( "%s(): blockIdx.x is %d. thread_index is %d\n", __func__, blockIdx.x, thread_index );
   conj_sqrs[thread_index] = hipCmulf( samples[thread_index], hipConjf( samples_d16[thread_index] ) );
}


__device__
void calc_conj_sqr_means( 
      hipfftComplex* __restrict__ conj_sqr_means, 
      const hipfftComplex* __restrict__ conj_sqrs, 
      const int conj_sqr_window_size, 
      const int num_vals 
   ) { 

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      hipfftComplex  t_conj_sqr_sum = make_hipFloatComplex(0.0,0.0);

      for( int w_index = 0; w_index < conj_sqr_window_size; ++w_index ) {
         t_conj_sqr_sum = hipCaddf( t_conj_sqr_sum, conj_sqrs[index + w_index] );
      }
      conj_sqr_means[index] = complex_divide_by_scalar( t_conj_sqr_sum, (float)conj_sqr_window_size );
   }

}


__device__
void calc_conj_sqr_means_smem( 
      hipfftComplex* __restrict__ conj_sqr_means, 
      const hipfftComplex* __restrict__ conj_sqrs, 
      const int conj_sqr_window_size, 
      const int num_vals 
   ) { 

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int thread_index = threadIdx.x;

   if ( thread_index < (blockDim.x - conj_sqr_window_size)  ) {
      //hipfftComplex  t_conj_sqr_sum = make_hipFloatComplex(0.0,0.0);
      conj_sqr_means[thread_index] = make_hipFloatComplex(0.0,0.0);

      for( int w_index = 0; w_index < conj_sqr_window_size; ++w_index ) {
         __syncthreads();
         //t_conj_sqr_sum = hipCaddf( t_conj_sqr_sum, conj_sqrs[thread_index + w_index] );
         conj_sqr_means[thread_index] = hipCaddf( conj_sqr_means[thread_index], conj_sqrs[thread_index + w_index] );
      }
      __syncthreads();
      //conj_sqr_means[thread_index] = complex_divide_by_scalar( t_conj_sqr_sum, (float)conj_sqr_window_size );
      conj_sqr_means[thread_index] = complex_divide_by_scalar( conj_sqr_means[thread_index], (float)conj_sqr_window_size );
   }
}


__device__
void calc_conj_sqr_mean_mags( float* __restrict__ conj_sqr_mean_mags, const hipfftComplex* __restrict__ conj_sqr_means, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      conj_sqr_mean_mags[index] = hipCabsf( conj_sqr_means[index] );
   }
}


__device__
void calc_conj_sqr_mean_magss( float* __restrict__ conj_sqr_mean_mags, const hipfftComplex* __restrict__ conj_sqr_means, const int num_vals ) {

   int thread_index = threadIdx.x;

   conj_sqr_mean_mags[thread_index] = hipCabsf( conj_sqr_means[thread_index] );
}


__device__
void calc_mag_sqrs( float* __restrict__ mag_sqrs, const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      float temp = hipCabsf( samples[index] );
      mag_sqrs[index] = temp * temp;
   }
}


__device__
void calc_mag_sqrss( float* __restrict__ mag_sqrs, const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int thread_index = threadIdx.x;

   float temp = hipCabsf( samples[thread_index] );
   mag_sqrs[thread_index] = temp * temp;
}


__device__
void calc_mag_sqr_means( 
      float* __restrict__ mag_sqr_means, 
      const float* __restrict__ mag_sqrs,
      const int mag_sqr_window_size, 
      const int num_vals 
   ) { 

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      float  t_mag_sqr_sum = 0.0;
      for( int w_index = 0; w_index < mag_sqr_window_size; ++w_index ) {
         t_mag_sqr_sum = t_mag_sqr_sum + mag_sqrs[index + w_index];
      }
      mag_sqr_means[index] = t_mag_sqr_sum/(float)mag_sqr_window_size;
   }

}


__device__
void normalize( float* __restrict__ norms, const float* __restrict__ conj_sqr_mean_mags, 
   const float* __restrict__ mag_sqr_means, const int num_samples ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_samples; index += stride) {
      if ( mag_sqr_means[index] > 0.f ) {
         norms[index] =  conj_sqr_mean_mags[index]/mag_sqr_means[index];
      } else {
         norms[index] = 0.f;
      }
   }
}


__device__
void normalizes( float* __restrict__ norms, const float* __restrict__ conj_sqr_mean_mags, 
   const float* __restrict__ mag_sqr_means, const int num_samples ) {

   int thread_index = threadIdx.x;

   if ( mag_sqr_means[thread_index] > 0.f ) {
      norms[thread_index] =  conj_sqr_mean_mags[thread_index]/mag_sqr_means[thread_index];
   } else {
      norms[thread_index] = 0.f;
   }
}


__global__
void norm_autocorr_kernel( 
   float* __restrict__ norms, 
   float* __restrict__ mag_sqr_means, 
   float* __restrict__ mag_sqrs, 
   float* __restrict__ conj_sqr_mean_mags, 
   hipfftComplex* __restrict__ conj_sqr_means, 
   hipfftComplex* __restrict__ conj_sqrs, 
   hipfftComplex* __restrict__ samples_d16, 
   const hipfftComplex* __restrict__ samples,
   const int conj_sqr_window_size, 
   const int mag_sqr_window_size,
   const int num_samples ) {

   int num_conj_sqr_sums = num_samples - conj_sqr_window_size;
   int num_mag_sqr_sums = num_samples - mag_sqr_window_size;

   __shared__ hipfftComplex s_samples[THREADS_PER_BLOCK];
   __shared__ hipfftComplex s_samples_d16[THREADS_PER_BLOCK];
   __shared__ hipfftComplex s_conj_sqrs[THREADS_PER_BLOCK];
   __shared__ hipfftComplex s_conj_sqr_means[THREADS_PER_BLOCK];
   __shared__ float s_conj_sqr_mean_mags[THREADS_PER_BLOCK];
   __shared__ float s_mag_sqrs[THREADS_PER_BLOCK];
   __shared__ float s_mag_sqr_means[THREADS_PER_BLOCK];
   __shared__ float s_norms[THREADS_PER_BLOCK];

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int thread_index = threadIdx.x;

   delay16<hipfftComplex>( samples_d16, samples, num_samples );
   __syncthreads();

   
}

