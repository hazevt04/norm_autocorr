#include "hip/hip_runtime.h"

#include "my_cufft_utils.hpp"

#include "norm_autocorr_kernel.cuh"

template<typename T>
__device__
void delay16( T* delayed_vals, const T* vals, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;

   if ( global_index < 16 ) {
      delayed_vals[global_index] = T{0};
   } else if ( global_index < num_vals ) {
      delayed_vals[global_index] = vals[global_index-16];
   }

}


template
__device__
void delay16<hipfftComplex>( hipfftComplex* delayed_vals, const hipfftComplex* vals, const int num_vals );


__device__
void auto_correlation( hipfftComplex* __restrict__ conj_sqrs, const hipfftComplex* __restrict__ samples_d16,
   const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      conj_sqrs[index] = hipCmulf( samples[index], hipConjf( samples_d16[index] ) );
   }
}


__device__
void complex_mag_squared( float* __restrict__ mag_sqrs, const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      float temp = hipCabsf( samples[index] );
      mag_sqrs[index] = temp * temp;
   }
}


__device__
void complex_mags( float* __restrict__ mags, const hipfftComplex* __restrict__ samples, const int num_vals ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      mags[index] = hipCabsf( samples[index] );
   }
}


__device__ __inline__
hipfftComplex complex_divide_by_scalar( hipfftComplex cval, float scalar_divisor ) {
   return make_hipFloatComplex( cval.x/scalar_divisor, cval.y/scalar_divisor );
}

__device__
void moving_averages( 
      hipfftComplex* __restrict__ conj_sqr_means, 
      float* __restrict__ mag_sqr_means, 
      const hipfftComplex* __restrict__ conj_sqrs, 
      const float* __restrict__ mag_sqrs,
      const int conj_sqr_window_size, 
      const int mag_sqr_window_size, 
      const int num_vals 
   ) { 

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_vals; index += stride) {
      hipfftComplex  t_conj_sqr_sum = make_hipFloatComplex(0.0,0.0);
      float  t_mag_sqr_sum = 0.0;
      for( int w_index = 0; w_index < conj_sqr_window_size; ++w_index ) {
         t_conj_sqr_sum = hipCaddf( t_conj_sqr_sum, conj_sqrs[index + w_index] );
      }
      for( int w_index = 0; w_index < mag_sqr_window_size; ++w_index ) {
         t_mag_sqr_sum = t_mag_sqr_sum + mag_sqrs[index + w_index];
      }
      conj_sqr_means[index] = complex_divide_by_scalar( t_conj_sqr_sum, (float)conj_sqr_window_size );
      mag_sqr_means[index] = t_mag_sqr_sum/(float)mag_sqr_window_size;
   }

}

__device__
void normalize( float* __restrict__ norms, const float* __restrict__ conj_sqr_mean_mags, 
   const float* __restrict__ mag_sqr_means, const int num_samples ) {

   int global_index = blockDim.x * blockIdx.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int index = global_index; index < num_samples; index += stride) {
      norms[index] =  conj_sqr_mean_mags[index]/mag_sqr_means[index];  
   }

}

__global__
void norm_autocorr_kernel( 
   float* __restrict__ norms, 
   float* __restrict__ mag_sqr_means, 
   float* __restrict__ mag_sqrs, 
   float* __restrict__ conj_sqr_mean_mags, 
   hipfftComplex* __restrict__ conj_sqr_means, 
   hipfftComplex* __restrict__ conj_sqrs, 
   hipfftComplex* __restrict__ samples_d16, 
   const hipfftComplex* __restrict__ samples,
   const int conj_sqr_window_size, 
   const int mag_sqr_window_size,
   const int num_samples ) {


   delay16<hipfftComplex>( samples_d16, samples, num_samples );
   auto_correlation( conj_sqrs, samples_d16, samples, num_samples );
   complex_mag_squared( mag_sqrs, samples, num_samples );
   moving_averages( conj_sqr_means, mag_sqr_means, conj_sqrs, mag_sqrs, 
      conj_sqr_window_size, mag_sqr_window_size, num_samples );
   complex_mags( conj_sqr_mean_mags, conj_sqr_means, num_samples );
   normalize( norms, conj_sqr_mean_mags, mag_sqr_means, num_samples );
}
